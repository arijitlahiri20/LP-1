
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#define N 1000
using namespace std;
using namespace std::chrono;

__global__ void ArraySum(float *array, float *sum){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < N)
        atomicAdd(sum, array[index]);

}

void findSum(float *array, float *sum){

    auto start = high_resolution_clock::now();
    for(int i = 0;i < N;i ++){
        *sum += array[i];
    }
    auto stop = high_resolution_clock::now();
    auto time_req = duration_cast<microseconds>(stop - start).count();
    cout << endl << "Sum from CPU is: " << *sum << endl;
    cout << endl << "Time required for CPU: " << time_req << endl;
}

int main(){

    float *hostInput, *deviceInput, *sumCPU, *sumGPU, *sumGPU2CPU;

    hostInput = (float*)malloc(N * sizeof(float));
    sumCPU = (float*)malloc(sizeof(float));
    sumGPU2CPU = (float*)malloc(sizeof(float));
    *sumCPU = 0;
    for(int i = 0;i < N;i ++){
        hostInput[i] = 1.0f;
    }

    hipMalloc(&deviceInput, N * sizeof(float));
    hipMalloc(&sumGPU, sizeof(float));

    hipMemcpy(deviceInput, hostInput, N * sizeof(float), hipMemcpyHostToDevice);

    findSum(hostInput, sumCPU);

    dim3 threadsPerBlock(512, 1, 1);
    dim3 numBlocks(512, 1, 1);

    auto start = high_resolution_clock::now();
    ArraySum<<<numBlocks, threadsPerBlock>>>(deviceInput, sumGPU);
    auto stop = high_resolution_clock::now();
    auto time_req = duration_cast<microseconds>(stop - start).count();

    hipMemcpy(sumGPU2CPU, sumGPU, sizeof(float), hipMemcpyDeviceToHost);

    cout << endl << "Sum from GPU is: " << *sumGPU2CPU << endl;
    cout << endl << "Time required for GPU: " << time_req << endl;

    free(hostInput);
    free(sumCPU);
    free(sumGPU2CPU);
    hipFree(deviceInput);
    hipFree(sumGPU);

}